#include "hip/hip_runtime.h"
/*
  CPE 471 Lab 1 
  Base code for Rasterizer
  Example code using B. Somers' image code - writes out a sample tga
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "glm/glm.hpp"
#include <math.h>
#include <algorithm>
#include "Image.h"
#include "types.h"
#include "VanExLib.h"


/*__constant__ Sphere s[NUM_SPHERES];
__constant__ Plane * f;
__constant__ PointLight * l;
__constant__ Camera *cam;*/

Camera* CameraInit();
PointLight* LightInit();
Sphere* CreateSpheres();
Plane* CreatePlanes();
__host__ __device__ Point CreatePoint(float x, float y, float z);
__host__ __device__ color_t CreateColor(float r, float g, float b);

__global__ void CUDARayTrace(Camera * cam, Plane * f, PointLight *l, Sphere * s, color_t * pixelList);
__global__ void CUDADummy(Camera * cam);//, Plane * f, PointLight *l, Sphere * s);

__device__ color_t RayTrace(Ray r, Sphere* s, Plane* f, PointLight* l);
__device__ color_t Shading(Ray r, Point p, Point normalVector, PointLight* l, color_t diffuse, color_t ambient, color_t specular); 
__device__ float SphereRayIntersection(Sphere* s, Ray r);
__device__ float PlaneRayIntersection(Plane* s, Ray r);
//__device__ float glm::dot(Point p1, Point p2);
//__device__ Point subtractPoints(Point p1, Point p2);
//__device__ Point glm::normalize(Point p);



/* 
 *  Handles CUDA errors, taking from provided sample code on clupo site
 */

static void HandleError( hipError_t err, const char * file, int line)
{
    if(err !=hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
            exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

int main(void) 
{
  // set up for random num generator
   //srand ( time(NULL) );
   srand ( 0 );
   Image img(WINDOW_WIDTH, WINDOW_HEIGHT);
   Camera* camera = CameraInit(), * cam_d;
   PointLight* light = LightInit(), *l_d ;
   color_t * pixel_device = NULL;
   float aspectRatio = WINDOW_WIDTH; 
   aspectRatio /= WINDOW_HEIGHT;
   hipEvent_t start, stop; 
   pixel_device = new color_t[WINDOW_WIDTH * WINDOW_HEIGHT];
  	
  	//SCENE SET UP
  	// (floor)
   Plane* floor = CreatePlanes(), *f_d;
   //floor->center = CreatePoint(0, -1 * WINDOW_HEIGHT / 2, -1 * WINDOW_WIDTH / 2);
   //floor->color = CreateColor(200, 200, 200);
   //floor->normal = CreatePoint(0, 0, -1 * WINDOW_WIDTH / 2);
   // (spheres)
   Sphere* spheres = CreateSpheres(), *s_d;


  // HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(cam), camera, sizeof(Camera)) );


   color_t * pixel_deviceD;
   HANDLE_ERROR( hipMalloc(&pixel_deviceD,sizeof(color_t) * WINDOW_WIDTH * WINDOW_HEIGHT) );

   HANDLE_ERROR( hipMalloc((void**)&cam_d, sizeof(Camera)) );
   HANDLE_ERROR( hipMalloc(&f_d, sizeof(Plane)) );
   HANDLE_ERROR( hipMalloc(&l_d, sizeof(PointLight)) );
   HANDLE_ERROR( hipMalloc(&s_d,  sizeof(Sphere)*NUM_SPHERES));
   
   HANDLE_ERROR( hipMemcpy(l_d, light, sizeof(PointLight), hipMemcpyHostToDevice) );
   HANDLE_ERROR( hipMemcpy(cam_d, camera,sizeof(Camera), hipMemcpyHostToDevice) );
   HANDLE_ERROR( hipMemcpy(f_d, floor,sizeof(Plane), hipMemcpyHostToDevice) );
   HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
   
   //CUDA Timing 
   HANDLE_ERROR( hipEventCreate(&start) );
   HANDLE_ERROR( hipEventCreate(&stop) );
   HANDLE_ERROR( hipEventRecord(start, 0));

   // The Kernel Call
   CUDARayTrace<<< (WINDOW_WIDTH * WINDOW_HEIGHT + 383) / 384, 384  >>>(cam_d, f_d, l_d, s_d, pixel_deviceD);
   //CUDARayTrace<<< 100, 575 >>>(cam_d, f_d, l_d, s_d, pixel_deviceD);

   //CUDADummy<<<1, 1>>>(cam_d);//, f_d, l_d, s_d);
   // Coming Back
   HANDLE_ERROR(hipEventRecord( stop, 0));
   HANDLE_ERROR(hipEventSynchronize( stop ));
   float elapsedTime;
   HANDLE_ERROR(hipEventElapsedTime( &elapsedTime, start, stop));

   printf("GPU computation time: %.1f ms\n", elapsedTime);

   HANDLE_ERROR( hipMemcpy(pixel_device, pixel_deviceD,sizeof(color_t) * WINDOW_WIDTH * WINDOW_HEIGHT, hipMemcpyDeviceToHost) );
   fflush(stdout);
   
   for (int i=0; i < WINDOW_WIDTH; i++) {
		for (int j=0; j < WINDOW_HEIGHT; j++) {
         //Looping over the Rays
     		img.pixel(i, j, pixel_device[j*WINDOW_WIDTH + i]);
		    }
  	}
  	
	// IMAGE OUTPUT
	//
  	// write the targa file to disk
  	img.WriteTga((char *)"raytraced.tga", true); 
  	// true to scale to max color, false to clamp to 1.0
   hipFree(pixel_deviceD);
} 


Camera* CameraInit() {
   
   Camera* c = new Camera();
   
   c->eye = CreatePoint(0, 0, 0);
   c->lookAt = CreatePoint(0, 0, SCREEN_DISTANCE);
   c->lookUp = CreatePoint(0, 1, 0);

   c->u = CreatePoint(1, 0, 0);
   c->v = CreatePoint(0, 1, 0);
   c->w = CreatePoint(0, 0, 1);
   
   return c;
}

PointLight* LightInit() {
   PointLight* l = new PointLight();

   l->ambient = CreateColor(0.2, 0.2, 0.2);
   l->diffuse = CreateColor(0.6, 0.6, 0.6);
   l->specular = CreateColor(0.4, 0.4, 0.4);

   l->position = CreatePoint(600, 0, -200);

   return l;
}

__host__  __device__ Point CreatePoint(float x, float y, float z) {
   Point p;
   
   p.x = x;
   p.y = y;
   p.z = z;

   return p;
}

__host__ __device__ color_t CreateColor(float r, float g, float b) {
   color_t c;

   c.r = r;
   c.g = g;
   c.b = b;
   c.f = 1.0;

   return c;
}

Sphere* CreateSpheres() {
   Sphere* spheres = new Sphere[NUM_SPHERES]();
   int i=0, j=0, k=0, num=0;

   while (num < NUM_SPHERES) {
      for (i=0; i < 6 && num < NUM_SPHERES; i++) {
         for (j=0; j < 5 && num < NUM_SPHERES; j++) {
            spheres[num].radius = 40. - rand() % 10;
            spheres[num].center = CreatePoint(WINDOW_WIDTH/10 + 100 - rand() % 300,
                                              100 - rand() % 200,
                                              -200. - rand() %200); //- rand() % 20);
            spheres[num].ambient = CreateColor(std::min((i + j) * .15, 1.),
                                               std::min((j + k) * .15, 1.),
                                               std::max(1. - (k + i) * .15, 0.));
            spheres[num].diffuse = CreateColor(std::min((i + j) * .15, 1.),
                                               std::min((j + k) * .15, 1.),
                                               std::max(1. - (k + i) * .15, 0.));
            spheres[num].specular = CreateColor(1., 1., 1.);
            num++;
         }
      }
      k++;
   }

   return spheres;

}

#define NUM_PLANES 1

Plane* CreatePlanes() {
   Plane* planes = new Plane[NUM_PLANES]();
   int num=0;
   while (num < NUM_PLANES) {
            planes[num].normal = CreatePoint(0,0,0) ;
            planes[num].center = CreatePoint(0,200,0);
            planes[num].ambient = CreateColor(.5,.5,.5);
            planes[num].diffuse =  CreateColor(.5,.5,.5);
            planes[num].specular = CreateColor(1.,1.,1.);
            num++;
   }
   return planes;
}

__global__ void CUDADummy(Camera * cam)//, Plane * f ,PointLight * l,Sphere * s)
{
  printf("C addr: %f\n", cam->lookAt.z);//, F addr: %f, L addr: %f, Sphere addr: %s", cam, f, l, s); 
}

__global__ void CUDARayTrace(Camera * cam,Plane * f,PointLight * l, Sphere * s, color_t * pixelList)
{
    float tanVal = tan(FOV/2);
    float aspectRatio = WINDOW_WIDTH / WINDOW_HEIGHT;
    int row = (blockIdx.x *blockDim.x + threadIdx.x ) / WINDOW_WIDTH;
    int col = (blockIdx.x *blockDim.x + threadIdx.x ) % WINDOW_WIDTH;
    color_t returnColor;
    Ray r;
    
    //BOUNDARY CHECK
    if(row > WINDOW_HEIGHT)
      return;

    //INIT RAY VALUES
	  r.origin = cam->eye;
    r.direction = cam->lookAt;
    r.direction.y = tanVal - (2 * tanVal / WINDOW_HEIGHT) * row;
    r.direction.x = -1 * aspectRatio * tanVal + (2 * tanVal / WINDOW_HEIGHT) * col;


    returnColor = RayTrace(r, s, f, l);
    int index = row *WINDOW_WIDTH + col;
    
    //if(index == 0)
    //  printf("I RAN, I WORKED\n");
    pixelList[index].r = returnColor.r;
    pixelList[index].g = returnColor.g;
    pixelList[index].b = returnColor.b;
    pixelList[index].f = returnColor.f;
    
}

__device__ color_t RayTrace(Ray r, Sphere* s, Plane* f, PointLight* l) {
   // color_t black = CreateColor(1.f, 0, 0); 
    float t, smallest;
   	Point p, normalVector;
   	int i = 0, closestSphere = -1, closestPlane = -1;

    
    while (i < NUM_SPHERES) {
      t = SphereRayIntersection(s + i, r);


      if (t > 0 && (closestSphere < 0 || t < smallest)) {
        smallest = t;
        closestSphere = i;
      }
      i++;
    }
i=0;
    while (i < NUM_PLANES) {
      t = PlaneRayIntersection(f, r);


      if (t > 0 && (closestSphere < 0 || t < smallest)) {
        smallest = t;
        closestPlane = i;
      }
      i++;
    }

    if(closestPlane > -1)
    {
      //plane closer than sphere
      p = CreatePoint(r.direction.x * smallest, r.direction.y * smallest, r.direction.z * smallest);
	    normalVector = glm::normalize(f[closestPlane].normal-f[closestPlane].center);
      return Shading(r, p, normalVector, l, f[closestPlane].diffuse,
      f[closestPlane].ambient,f[closestPlane].specular);
    }

    else   if (closestSphere > -1) {
      p = CreatePoint(r.direction.x * smallest, r.direction.y * smallest, r.direction.z * smallest);
	    normalVector = glm::normalize((s[closestSphere].center)-p);
      return Shading(r, p, normalVector, l, s[closestSphere].diffuse,
      s[closestSphere].ambient,s[closestSphere].specular);
    }

    return CreateColor(0,0,0);
}

__device__ float PlaneRayIntersection(Plane *p, Ray r)
{
  float t;
  Point N = p->normal - p->center;
  float denominator = glm::dot(r.direction,N);
  if(denominator!=0)
  {
    t = (glm::dot(p->center-r.origin,N)) / denominator;
    return t;
  }
  else
  {
    return -1;
  }
}


__device__ float SphereRayIntersection(Sphere* s, Ray r) {
	float a, b, c, d, t1, t2;
    
    a = glm::dot((r.direction), (r.direction));
    b = glm::dot((r.origin)- (s->center),(r.direction));
    c = glm::dot((r.origin)-(s->center), (r.origin)- (s->center))
            - (s->radius * s->radius);
    d = (b * b) - (a * c);
    
    if (d >= 0) {

		t1 = (-1 * b - sqrt(d)) / a;
		t2 = (-1 * b + sqrt(d)) / a;
    
		if (t2 < t1 && t2 > 0) {
			return t2;
		} else if (t1 > 0) {
			return t1;
		}
	}
	return d;
}

__device__ color_t Shading(Ray r, Point p, Point normalVector,
PointLight* l, color_t diffuse, color_t ambient, color_t specular) {
	color_t a, d, s, total;
	float NdotL, RdotV;
	Point viewVector, lightVector, reflectVector;

   //printf("r->%lf g->%lf b->%lf\n", l->ambient->r, l->ambient->g, l->ambient->b);
   //printf("r->%lf g->%lf b->%lf\n", l->diffuse->r, l->diffuse->g, l->diffuse->b);
   //printf("r->%lf g->%lf b->%lf\n\n", l->specular->r, l->specular->g, l->specular->b);

	viewVector = glm::normalize((r.origin)-p);
	
	lightVector = glm::normalize((l->position) -p);
	
  NdotL = glm::dot(lightVector, normalVector);
//  reflectVector = normalVector - lightVector;
  reflectVector = (2.f *normalVector*NdotL) -lightVector;
 // reflectVector = glm::reflect(-lightVector,normalVector);
	/*
  reflectTemp = 2 * NdotL;
	reflectVector.x *= reflectTemp;
	reflectVector.y *= reflectTemp;
	reflectVector.z *= reflectTemp;
	*/

  a.r = l->ambient.r * ambient.r;
	a.g = l->ambient.g * ambient.g;
	a.b = l->ambient.b * ambient.b;
  
  // Diffuse
  d.r = NdotL * l->diffuse.r * diffuse.r * (NdotL > 0);
  d.g = NdotL * l->diffuse.g * diffuse.g * (NdotL > 0);
  d.b = NdotL * l->diffuse.b * diffuse.b * (NdotL > 0);
      
  // Specular
  RdotV = glm::pow(glm::dot(glm::normalize(reflectVector), viewVector), 100.f);
  //RdotV = glm::dot(reflectVector,viewVector) *glm::dot(reflectVector,viewVector) ;
  s.r = RdotV * l->specular.r * specular.r * (NdotL > 0) *(RdotV>0);
  s.g = RdotV * l->specular.g * specular.g * (NdotL > 0) *(RdotV>0);
  s.b = RdotV * l->specular.b * specular.b * (NdotL > 0) *(RdotV>0);
/*	
  total.r =  -s.r;
	total.g =  -s.g;
	total.b =  -s.b;*/
  total.r = glm::min(1.f, a.r + d.r + s.r);
	total.g = glm::min(1.f, a.g + d.g + s.g);
	total.b = glm::min(1.f, a.b + d.b + s.b);
  total.f = 1.f;
	return total;
}
/*
__device__ Point glm::normalize(Point p) {
	float d = sqrt(glm::dot(p, p));
  
  p.x /= d;
	p.y /= d;
	p.z /= d;
	
	return p;
}
*/
/*
__device__ float glm::dot(Point p1, Point p2) {
  return p1.x * p2.x + p1.y * p2.y + p1.z * p2.z;
//	return glm::dot(p1,p2);
}
*/
// This is essentially p1 - p2:
/*
__device__ Point subtractPoints(Point p1, Point p2) {
   Point p3;

   p3.x = p1.x - p2.x;
   p3.y = p1.y - p2.y;
   p3.z = p1.z - p2.z;
   
  return p1-p2;

}*/
