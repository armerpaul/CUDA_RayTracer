#include "hip/hip_runtime.h"
/*
 * CPE 570 && CPE 458 Duet
 * Ray Tracer
 * Professor Christopher Lupo and Professor Zoe" Wood
 * Paul Armer(parmer), Bryan Ching(bcching), Matt Crussell(macrusse)
 */

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "glm/glm.hpp"
#include <math.h>
#include <algorithm>
#include "Image.h"
#include "types.h"
#include "cudaRayTrace.h"

#include "glm/gtc/matrix_transform.hpp"
#include "glm/gtx/rotate_vector.hpp"

Camera * camera, *cam_d;
PointLight *light, *l_d;
Plane * planes, *p_d;
Sphere * spheres, *s_d;
float theta;

Camera* CameraInit();
PointLight* LightInit();
Sphere* CreateSpheres();
Plane* CreatePlanes();
__host__ __device__ Point CreatePoint(float x, float y, float z);
__host__ __device__ color_t CreateColor(float r, float g, float b);

__global__ void CUDARayTrace(Camera * cam, Plane * f, PointLight *l, Sphere * s, uchar4 * position);

__device__ color_t RayTrace(Ray r, Sphere* s, Plane* f, PointLight* l);
__device__ color_t SphereShading(int sNdx, Ray r, Point p, Sphere* sphereList, PointLight* l);
__device__ color_t Shading(Ray r, Point p, Point normalVector, PointLight* l, color_t diffuse, color_t ambient, color_t specular); 
__device__ float SphereRayIntersection(Sphere* s, Ray r);
__device__ float PlaneRayIntersection(Plane* s, Ray r);

static void HandleError( hipError_t err, const char * file, int line)
{
    if(err !=hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
            exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

/* 
 *  Handles CUDA errors, taking from provided sample code on clupo site
 */
extern "C" void setup_scene()
{
	 HANDLE_ERROR( hipDeviceSetCacheConfig(hipFuncCachePreferL1));
   camera = CameraInit();
   light = LightInit();
   spheres = CreateSpheres();
   planes = CreatePlanes(); 
   HANDLE_ERROR( hipMalloc((void**)&cam_d, sizeof(Camera)) );
   HANDLE_ERROR( hipMalloc(&p_d, sizeof(Plane)*NUM_PLANES) );
   HANDLE_ERROR( hipMalloc(&l_d, sizeof(PointLight)) );
   HANDLE_ERROR( hipMalloc(&s_d,  sizeof(Sphere)*NUM_SPHERES));
   

   HANDLE_ERROR( hipMemcpy(l_d, light, sizeof(PointLight), hipMemcpyHostToDevice) );

   HANDLE_ERROR( hipMemcpy(cam_d, camera,sizeof(Camera), hipMemcpyHostToDevice) );
   HANDLE_ERROR( hipMemcpy(p_d, planes,sizeof(Plane)*NUM_PLANES, hipMemcpyHostToDevice) );


   HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
   theta = 0;
}

extern "C" void ijklMove(unsigned char key)
{
   glm::mat4 mvp = glm::lookAt(camera->eye, camera->lookAt, camera->lookUp);
   //GLfloat angle = (GLFloat)(10);
   //glm::mat4 View = glm::mat4(1.);
//   View = glm::rotate(View, angle * -1.0f, glm::vec3(1.f, 0.f, 0.f));
   //View = glm::rotate(View, angle * 0.5f, glm::vec3(0.f, 1.f, 0.f));
//   View = glm::rotate(View, angle * 0.5f, glm::vec3(0.f, 0.f, 1.f)); 
  //mvp = view * mvp;*/
/*  float sin_theta_x, cos_theta_x, sin_theta_y,cos_theta_y;

  switch(key){
    case('i'):
      camera->theta_x+=.01;
      break; 
    case('k'):
      camera->theta_x-=.01;
      break;
    case('j'):
      camera->theta_y+=.01;
      break;
    case('l'):
      camera->theta_y-=.01;
      break;
   }
   sin_theta_x = sin(camera->theta_x);
   sin_theta_y = sin(camera->theta_y);
   cos_theta_x = cos(camera->theta_x);
   cos_theta_y = cos(camera->theta_y);

   camera->lookAt = camera->eye + CreatePoint(sin_theta_y,cos_theta_x, sin_theta_x - cos_theta_y - 1);
   camera->lookUp = camera->eye + CreatePoint(0,sin_theta_x + 1, -1 * cos_theta_x);
   camera->lookRight = camera->eye + CreatePoint(1 + cos_theta_y , 0 ,
   sin_theta_y);*/
}



extern "C" void wasdMove(unsigned char key)
{

   Point move;
   switch(key){
    case('w'):
      move = .01f * glm::normalize(camera->lookAt - camera->eye);
      break; 
    case('s'):
      move = .01f * glm::normalize(camera->eye - camera->lookAt);
      break;
    case('a'):
      move = .01f * glm::normalize(camera->eye - camera->lookRight);
      break;
    case('d'):
      move = .01f * glm::normalize(camera->lookRight - camera->eye);
      break;
   }
   camera->lookAt += move;
   camera->lookUp += move;
   camera->lookRight += move;
   camera->eye += move;
}
extern "C" void launch_kernel(uchar4* pos, unsigned int image_width, 
                  unsigned int image_height, float time)
{
  
  // set up for random num generator
  // srand ( time(NULL) );
   
   hipEvent_t start, stop; 
   Point move;

 //  move.y = .001 * sin(theta += .01);
 //  move.x = .001 * cos(theta);
 //  move.z += .0001;
   light->position.x -= 2 *sin(theta += .01);	

 //  camera->lookAt += move;
   //camera->lookUp += move;
   //camera->eye += move;
   //SCENE SET UP

   
   spheres[NUM_SPHERES-1].radius=5;
   spheres[NUM_SPHERES-1].center=light->position;
   spheres[NUM_SPHERES-1].ambient=CreateColor(1,0,0);
   spheres[NUM_SPHERES-1].diffuse=CreateColor(1,1,1);
   spheres[NUM_SPHERES-1].specular=CreateColor(1,1,1);
   
   HANDLE_ERROR( hipMemcpy(l_d, light, sizeof(PointLight), hipMemcpyHostToDevice) );

   HANDLE_ERROR( hipMemcpy(cam_d, camera,sizeof(Camera), hipMemcpyHostToDevice) );
   
   HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
   //CUDA Timing 
   HANDLE_ERROR( hipEventCreate(&start) );
   HANDLE_ERROR( hipEventCreate(&stop) );
   HANDLE_ERROR( hipEventRecord(start, 0));

   // The Kernel Call
   dim3 gridSize((WINDOW_WIDTH+15)/16, (WINDOW_HEIGHT+15)/16);
   dim3 blockSize(16,16);
   CUDARayTrace<<< gridSize, blockSize  >>>(cam_d, p_d, l_d, s_d, pos);
hipDeviceSynchronize();
   // Coming Back
   HANDLE_ERROR(hipEventRecord( stop, 0));
   HANDLE_ERROR(hipEventSynchronize( stop ));
   float elapsedTime;
   HANDLE_ERROR(hipEventElapsedTime( &elapsedTime, start, stop));

   //printf("GPU computation time: %.1f ms\n", elapsedTime);

   //HANDLE_ERROR( hipMemcpy(pixel_device, pixel_deviceD,sizeof(color_t) * WINDOW_WIDTH * WINDOW_HEIGHT, hipMemcpyDeviceToHost) );
   //fflush(stdout);

   // hipFree(pixel_deviceD);
} 

/*
 * Initializes camera at point (X,Y,Z)
 */
Camera* CameraInit() {
   
   Camera* c = new Camera();
   
   c->eye = CreatePoint(0, 0, 0);//(X,Y,Z)
   c->lookAt = CreatePoint(0, 0, SCREEN_DISTANCE);
   c->lookUp = CreatePoint(0, 1, 0);
   c->lookRight = CreatePoint(1, 0, 0);
   c->theta_x = 0;
   c->theta_y = 0;
   return c;
}
/*
 * Initializes light at hardcoded (X,Y,Z) coordinates
 */
PointLight* LightInit() {
   PointLight* l = new PointLight();

   l->ambient = CreateColor(0.2, 0.2, 0.2);
   l->diffuse = CreateColor(0.6, 0.6, 0.6);
   l->specular = CreateColor(0.4, 0.4, 0.4);

   l->position = CreatePoint(50, 50, -300);

   return l;
}
/*
 * Creates a point, for GLM Point has been defined as vec3
 */
__host__  __device__ Point CreatePoint(float x, float y, float z) {
   Point p;
   
   p.x = x;
   p.y = y;
   p.z = z;

   return p;
}
/*
 * Creates a color_t type color based on input values
 */
__host__ __device__ color_t CreateColor(float r, float g, float b) {
   color_t c;

   c.r = r;
   c.g = g;
   c.b = b;
   c.f = 1.0;

   return c;
}
/*
 * Creates NUM_SPHERES # of Spheres, with randomly chosen values on color, location, and size
 */
Sphere* CreateSpheres() {
   Sphere* spheres = new Sphere[NUM_SPHERES]();
   float randr, randg, randb;
   int num = 0;
   while (num < NUM_SPHERES-1) {
            randr = (rand()%1000) /1000.f ;
            randg = (rand()%1000) /1000.f ;
            randb = (rand()%1000) /1000.f ;
            spheres[num].radius = 11. - rand() % 10;
            spheres[num].center = CreatePoint(-100 + rand() % 200,
                                              100 - rand() % 200,
                                              -200. - rand() %200);
            spheres[num].ambient = CreateColor(randr, randg, randb);
            spheres[num].diffuse = CreateColor(randr, randg, randb);
            spheres[num].specular = CreateColor(1., 1., 1.);
            num++;
   }
   

   spheres[NUM_SPHERES-1].radius=5;
   spheres[NUM_SPHERES-1].center=light->position;
   spheres[NUM_SPHERES-1].ambient=CreateColor(1,0,0);
   spheres[NUM_SPHERES-1].diffuse=CreateColor(1,1,1);
   spheres[NUM_SPHERES-1].specular=CreateColor(1,1,1);

   return spheres;

}
/*
 * CUDA global function which performs ray tracing. Responsible for initializing and writing to output vector
 */


Plane* CreatePlanes() {
   Plane* planes = new Plane[NUM_PLANES]();
   int num=0;
   /*while (num < NUM_PLANES) {

            planes[num].normal = CreatePoint(0,0,0) ;
            planes[num].center = CreatePoint(0,0,-500);
            planes[num].ambient = CreateColor(.5,.5,.5);
            planes[num].diffuse =  CreateColor(.5,.5,.5);
            planes[num].specular = CreateColor(1.,1.,1.);
            num++;
   }*/

           
            planes[0].normal = CreatePoint(0,0,0) ;
            planes[0].center = CreatePoint(0,0,-1000);
            planes[0].ambient = CreateColor(1,0,0);
            planes[0].diffuse =  CreateColor(1,0,0);
            planes[0].specular = CreateColor(1,0,0);

            planes[3].normal = CreatePoint(100,-100,0) ;
            planes[3].center = CreatePoint(500,-500,-500);
            planes[3].ambient = CreateColor(0,1,0);
            planes[3].diffuse =  CreateColor(0,1,0);
            planes[3].specular = CreateColor(0.,1.,0);

            planes[2].normal = CreatePoint(-100,100,0) ;
            planes[2].center = CreatePoint(-500,500,-500);
            planes[2].ambient = CreateColor(0,0,1);
            planes[2].diffuse =  CreateColor(0,0,1);
            planes[2].specular = CreateColor(0,0,1);

            planes[1].normal = CreatePoint(100,100,0) ;
            planes[1].center = CreatePoint(500,500,-500);
            planes[1].ambient = CreateColor(1,1,0);
            planes[1].diffuse =  CreateColor(1,1,0);
            planes[1].specular = CreateColor(1,1,0);

            planes[4].normal = CreatePoint(-100,-100,0) ;
            planes[4].center = CreatePoint(-500,-500, -500);
            planes[4].ambient = CreateColor(1,0,1);
            planes[4].diffuse =  CreateColor(1,0,1);
            planes[4].specular = CreateColor(1.,0,1.);
   return planes;
}
__global__ void CUDARayTrace(Camera * cam,Plane * f,PointLight * l, Sphere * s, uchar4 * pos)
{
    float tanVal = tan(FOV/2);

    //CALCULATE ABSOLUTE ROW,COL
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    color_t returnColor;
    Ray r;
    
    //BOUNDARY CHECK
    if(row >= WINDOW_HEIGHT || col >= WINDOW_WIDTH)
      return;

    //INIT RAY VALUES
	  r.origin = cam->eye;
    r.direction = cam->lookAt;
    r.direction.y += tanVal - (2 * tanVal / WINDOW_HEIGHT) * row;
    r.direction.x += -1 * WINDOW_WIDTH / WINDOW_HEIGHT * tanVal + (2 * tanVal / WINDOW_HEIGHT) * col;


    //RAY TRACE
    returnColor = RayTrace(r, s, f, l);
    
    //CALC OUTPUT INDEX
    int index = row *WINDOW_WIDTH + col;
    
    //PLACE DATA IN INDEX
    pos[index].x = 0xFF * returnColor.r ;
    pos[index].y = 0xFF * returnColor.g;
    pos[index].z = 0xFF * returnColor.b;
    pos[index].w = 0xFF * returnColor.f;
    
}
/*
 * Performs Ray tracing over all spheres for any ray r
 */
__device__ color_t RayTrace(Ray r, Sphere* s, Plane* f, PointLight* l) {
    color_t color = CreateColor(0, 0, 0); 
    float t, smallest;
   	int i = 0, closestSphere = -1, closestPlane = -1,  inShadow = false;
    Point normalVector;
    //FIND CLOSEST SPHERE ALONG RAY R
    while (i < NUM_SPHERES) {
      t = SphereRayIntersection(s + i, r);

      if (t > 0 && (closestSphere < 0 || t < smallest)) {
        smallest = t;
			  closestSphere = i;
		  }
      i++;
    }
    
i=0;
    while (i < NUM_PLANES) {
      t = PlaneRayIntersection(f + i, r);
      if (t > 0 && ( (closestSphere < 0 && closestPlane < 0) || t < smallest)) {//POSSIBLE LOGIC FIX CLOSESTSPHERE >1
        smallest = t;
        closestSphere = -1;
        closestPlane = i;
      }
      i++;
   }
    
    //SETUP FOR SHADOW CALCULATIONS
    i = 0;
    Ray shadowRay;
    shadowRay.origin = CreatePoint(r.direction.x * smallest, r.direction.y * smallest, r.direction.z * smallest);

    shadowRay.direction = l->position - shadowRay.origin;
   
    //DETERMINE IF SPHERE IS BLOCKING RAY FROM LIGHT TO SPHERE
    if(closestSphere > -1 || closestPlane > -1)
    {
      while (i <NUM_SPHERES-1 && !inShadow){ 
        t = SphereRayIntersection(s + i, shadowRay);
        if(i != closestSphere && t < 1 && t > 0){
//	printf("%f\n",t);
          inShadow = true;
        }
        i++;
      }
      i = 0;
      while(i < NUM_PLANES && !inShadow){
        t = PlaneRayIntersection(f + i, shadowRay);
        if(i != closestPlane && t < 1 && t > 0){
          inShadow = true;
        }
        i++;
      }
    }
   //inShadow = false; 
    if(closestPlane > -1 && !inShadow)
    {
      //plane closer than sphere
	    normalVector = glm::normalize(f[closestPlane].normal-f[closestPlane].center);
      return Shading(r, shadowRay.origin, normalVector, l, f[closestPlane].diffuse,
      f[closestPlane].ambient,f[closestPlane].specular);
    }
    if(closestPlane > -1)
    {
      color.r = l->ambient.r * f[closestPlane].ambient.r;
      color.g = l->ambient.g * f[closestPlane].ambient.g;
      color.b = l->ambient.b * f[closestPlane].ambient.b;
      //return CreateColor(1,1,1);
      return color;
    }

    //IF SHADOWED, ONLY SHOW AMBIENT LIGHTING
    if(closestSphere > -1 && !inShadow)
    {

	    normalVector = glm::normalize(shadowRay.origin-(s[closestSphere].center));
      return Shading(r, shadowRay.origin, normalVector, l, s[closestSphere].diffuse,
      s[closestSphere].ambient,s[closestSphere].specular);
    }
    if(closestSphere > -1)
    {
      color.r = l->ambient.r * s[closestSphere].ambient.r;
      color.g = l->ambient.g * s[closestSphere].ambient.g;
      color.b = l->ambient.b * s[closestSphere].ambient.b;
    }
    return color;
}

__device__ float PlaneRayIntersection(Plane *p, Ray r)
{
  float t;
  Point N = p->normal - p->center;
  float denominator = glm::dot(r.direction,N);
  if(denominator!=0)
  {
    t = (glm::dot(p->center-r.origin,N)) / denominator;
    if (t>100000)
      return -1;
    return t;
//    return glm::min(t,100000.f);
  }
  else
  {
    return -1;
  }
}


/*
 * Determines distance of intersection of Ray with Sphere, -1 returned if no intersection
 */
__device__ float SphereRayIntersection(Sphere* s, Ray r) {
	  float a, b, c, d, t1, t2;
    
    a = glm::dot((r.direction), (r.direction));
    b = glm::dot((r.origin)- (s->center),(r.direction));
    c = glm::dot((r.origin)-(s->center), (r.origin)- (s->center))
            - (s->radius * s->radius);
    d = (b * b) - (a * c);
    
    if (d >= 0) {

		t1 = (-1 * b - sqrt(d)) / (a);
		t2 = (-1 * b + sqrt(d)) / (a);
    
    if (t2 > t1 && t1 > 0) {
			return t1;
		
    } else if (t2 > 0) {
			return t2;
		
    }
	}
	return -1;
}
__device__ color_t Shading(Ray r, Point p, Point normalVector,
PointLight* l, color_t diffuse, color_t ambient, color_t specular) {
	color_t a, d, s, total;
	float NdotL, RdotV;
	Point viewVector, lightVector, reflectVector;

   //printf("r->%lf g->%lf b->%lf\n", l->ambient->r, l->ambient->g, l->ambient->b);
   //printf("r->%lf g->%lf b->%lf\n", l->diffuse->r, l->diffuse->g, l->diffuse->b);
   //printf("r->%lf g->%lf b->%lf\n\n", l->specular->r, l->specular->g, l->specular->b);

	viewVector = glm::normalize((r.origin)-p);
	
	lightVector = glm::normalize((l->position) -p);
	
  NdotL = glm::dot(lightVector, normalVector);
//  reflectVector = normalVector - lightVector;
  reflectVector = (2.f *normalVector*NdotL) -lightVector;
 // reflectVector = glm::reflect(-lightVector,normalVector);
	/*
  reflectTemp = 2 * NdotL;
	reflectVector.x *= reflectTemp;
	reflectVector.y *= reflectTemp;
	reflectVector.z *= reflectTemp;
	*/

  a.r = l->ambient.r * ambient.r;
	a.g = l->ambient.g * ambient.g;
	a.b = l->ambient.b * ambient.b;
  
  // Diffuse
  d.r = NdotL * l->diffuse.r * diffuse.r * (NdotL > 0);
  d.g = NdotL * l->diffuse.g * diffuse.g * (NdotL > 0);
  d.b = NdotL * l->diffuse.b * diffuse.b * (NdotL > 0);
      
  // Specular
  RdotV = glm::pow(glm::dot(glm::normalize(reflectVector), viewVector), 100.f);
  //RdotV = glm::dot(reflectVector,viewVector) *glm::dot(reflectVector,viewVector) ;
  s.r = RdotV * l->specular.r * specular.r * (NdotL > 0) *(RdotV>0);
  s.g = RdotV * l->specular.g * specular.g * (NdotL > 0) *(RdotV>0);
  s.b = RdotV * l->specular.b * specular.b * (NdotL > 0) *(RdotV>0);
/*	
  total.r =  -s.r;
	total.g =  -s.g;
	total.b =  -s.b;*/
  total.r = glm::min(1.f, a.r + d.r + s.r);
	total.g = glm::min(1.f, a.g + d.g + s.g);
	total.b = glm::min(1.f, a.b + d.b + s.b);
  total.f = 1.f;
	return total;
}
/*
 * Calculates Ambient, Diffuse, and Specular Shading for a single Ray
 */
__device__ color_t SphereShading(int sNdx, Ray r, Point p, Sphere* sphereList, PointLight* l) {
	  color_t a, d, s, total;
	  float NdotL, RdotV;
	  Point viewVector, lightVector, reflectVector, normalVector;

	  viewVector = glm::normalize((r.origin)-p);
	
	  lightVector = glm::normalize((l->position) -p);
	  normalVector = glm::normalize(p-(sphereList[sNdx].center));
	
    NdotL = glm::dot(lightVector, normalVector);
    reflectVector = (2.f *normalVector*NdotL)-lightVector;

    // Ambient
    a.r = l->ambient.r * sphereList[sNdx].ambient.r;
	  a.g = l->ambient.g * sphereList[sNdx].ambient.g;
	  a.b = l->ambient.b * sphereList[sNdx].ambient.b;
  
    // Diffuse
    d.r = NdotL * l->diffuse.r * sphereList[sNdx].diffuse.r * (NdotL > 0);
    d.g = NdotL * l->diffuse.g * sphereList[sNdx].diffuse.g * (NdotL > 0);
    d.b = NdotL * l->diffuse.b * sphereList[sNdx].diffuse.b * (NdotL > 0);
      
    // Specular
    RdotV = glm::pow(glm::dot(glm::normalize(reflectVector), viewVector), 100.f);
    s.r = RdotV * l->specular.r * sphereList[sNdx].specular.r * (NdotL > 0);
    s.g = RdotV * l->specular.g * sphereList[sNdx].specular.g * (NdotL > 0);
    s.b = RdotV * l->specular.b * sphereList[sNdx].specular.b * (NdotL > 0);
  
    total.r = glm::min(1.f, a.r + d.r+ s.r);
	  total.g = glm::min(1.f, a.g + d.g+ s.g);
	  total.b = glm::min(1.f, a.b + d.b+ s.b);
    total.f = 1.f;
	  
    return total;
}
